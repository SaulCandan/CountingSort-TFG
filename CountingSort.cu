#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "math.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

void mostrarArray(int* array, int tamanho) {

    int* guardado = array;
    for (int i = 0; i < tamanho; i++) {
        printf("%d, ", *guardado++);
    }
    printf("\n");
}

int* generarArrayAleatorio(int n, int min, int max) {
    int* array = (int*)malloc(n * sizeof(int));
    if (array == NULL) {
        printf("Error al asignar memoria para el array.\n");
        return NULL;
    }

    for (int i = 0; i < n; i++) {
        array[i] = min + (int)(((double)rand() / RAND_MAX) * (max - min + 1));
    }

    return array;
}

__global__ void tester1OrdenadoParalelokernel(int* sorted_device, int sortedSize, int* first_last_device, int first_last_Size, int nHilosUsados, bool* toret) {
    int idHilo = threadIdx.x + blockIdx.x * blockDim.x;
    int nHilosTotal = blockDim.x * gridDim.x;
    if (idHilo < nHilosUsados)
    {

        int inicio, fin;
        int nposRecorrer = (sortedSize / nHilosUsados);

        inicio = idHilo * nposRecorrer;
        fin = (idHilo * nposRecorrer) + nposRecorrer;

        if (idHilo == nHilosUsados - 1) { fin += (sortedSize % nHilosUsados); }

        for (int i = inicio; i < fin - 1; i++)
        {
            if (sorted_device[i] > sorted_device[i + 1]) { *toret = false; }
        }
        first_last_device[(idHilo * 2)] = sorted_device[inicio];
        first_last_device[(idHilo * 2) + 1] = sorted_device[fin - 1];

    }

}

__global__ void tester2OrdenadoParalelokernel( int* first_last_device, int first_last_Size, bool* toret) {
    int idHilo = threadIdx.x + blockIdx.x * blockDim.x;
    int nHilosTotal = blockDim.x * gridDim.x;
    
    if (idHilo == 0)
    {
        for (int i = 0; i < first_last_Size - 1; i++)
        {
            if (first_last_device[i] > first_last_device[i + 1]) { *toret = false; }
        }
    }

}

int testerOrdenadoParalelo(int* sorted_device, int sortedSize, int numBloques, int numHilos) {
    int nHilosUsados;
    if (numBloques * numHilos > sortedSize) {
        nHilosUsados = sortedSize;
    }
    else {

        nHilosUsados = numBloques * numHilos;
    }

    int* first_last_device;
    int first_last_Size = nHilosUsados * 2;
    hipError_t cudaStatusMalloc = hipMalloc(&first_last_device, first_last_Size * sizeof(int));
    if (cudaStatusMalloc != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (Testeando)");
        hipFree(first_last_device);
        return 2;
    }
    hipError_t cudaStatusMemcpy;

    bool verdad = true;
    bool* ordenado;
    cudaStatusMalloc = hipMalloc(&ordenado, sizeof(bool));
    if (cudaStatusMalloc != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (Testeando)");
        hipFree(ordenado);
        return 2;
    }
    cudaStatusMemcpy = hipMemcpy(ordenado, &verdad, sizeof(bool), hipMemcpyHostToDevice);
    if (cudaStatusMemcpy != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! (Array: verdad)");
        hipFree(&verdad);
        return 2;
    }

    tester1OrdenadoParalelokernel << <numBloques, numHilos >> > (sorted_device, sortedSize, first_last_device, first_last_Size, nHilosUsados, ordenado);
    hipDeviceSynchronize();
    tester2OrdenadoParalelokernel << <numBloques, numHilos >> > (first_last_device, first_last_Size, ordenado);
    hipDeviceSynchronize();

    bool* ordenadoResult = (bool*)malloc(sizeof(bool));
    cudaStatusMemcpy = hipMemcpy(ordenadoResult, ordenado, sizeof(bool), hipMemcpyDeviceToHost);
    if (cudaStatusMemcpy != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! (Array: ordenado)");
        hipFree(ordenado);
        return 2;
    }


    hipFree(ordenado);
    hipFree(first_last_device);

    if (*ordenadoResult) {
        return 0;
    }
    else {
        return 1;
    }

}

void presentacion() {
    int ndisp = 0;
    hipError_t errorcudaGetDevice = hipGetDevice(&ndisp);
    if (errorcudaGetDevice != hipSuccess) {
        fprintf(stderr, "hipGetDevice failed!\n");
        printf("Error en la presentacion\n");
    }
    else {
        hipDeviceProp_t propiedades;
        hipDeviceProp_t* propiedadesPuntero = &propiedades;
        hipGetDeviceProperties(propiedadesPuntero, ndisp);
        printf("Usando la grafica: %s\n", propiedades.name);
        //printf("Numero de multiprocesadores: %d\n", propiedades.multiProcessorCount);
        printf("Maximo de bloques por multiprocesador : %d\n", propiedades.maxBlocksPerMultiProcessor);
        printf("Maximo de hilos por bloque: %d\n", propiedades.maxThreadsPerBlock);
        printf("Maximo de hilos posibles: %d\n", propiedades.maxThreadsPerMultiProcessor);
        printf("Warp: %d\n", propiedades.warpSize);
        size_t free_mem, total_mem;
        hipMemGetInfo(&free_mem, &total_mem);
        printf("Memoria disponible en la GPU: %zu bytes\n\n", free_mem);
    }
}

//METODOS USADOS POR EL METODO PRINCIPAL COUNTING SORT
__global__ void findMinMax(int* array, int size, int* min, int* max, int* findmin, int* findmax) {

    int idHilo = threadIdx.x + blockIdx.x * blockDim.x;
    int nHilosTotal = blockDim.x * gridDim.x;

    int nHilosUsados;

    if (nHilosTotal > size)
    {
        nHilosUsados = size;
    }
    else {
        nHilosUsados = nHilosTotal;
    }

    if (idHilo < nHilosUsados)
    {
        int inicio, fin;
        int nposRecorrer = (size / nHilosUsados);

        inicio = idHilo * nposRecorrer;
        fin = (idHilo * nposRecorrer) + nposRecorrer;
        if (idHilo == nHilosUsados - 1) { fin += (size % nHilosUsados); }
        int localmin, localmax;


        localmin = array[inicio];
        localmax = array[inicio];
        for (int i = inicio; i < fin; i++)
        {
            if (localmin > array[i]) { localmin = array[i]; }
            if (localmax < array[i]) { localmax = array[i]; }
        }

        findmin[idHilo] = localmin;
        findmax[idHilo] = localmax;
    }
}

__global__ void calcularMinMax(int size, int* min, int* max, int* findmin, int* findmax) {
    int idHilo = threadIdx.x + blockIdx.x * blockDim.x;
    int nHilosTotal = blockDim.x * gridDim.x;

    int nHilosUsados;

    if (nHilosTotal > size)
    {
        nHilosUsados = size;
    }
    else {
        nHilosUsados = nHilosTotal;
    }

    if (idHilo == 0) {
        *min = findmin[0];
        *max = findmax[0];
        for (int i = 1; i < nHilosUsados; i++)
        {
            if (*min > findmin[i]) { *min = findmin[i]; }
            if (*max < findmax[i]) { *max = findmax[i]; }
        }
    }

}

int findMinMaxDeInput(int* input_device, int inputSize, int* minimo, int* maximo, int numBloques, int numHilos) {

    int* min_device;
    hipMalloc(&min_device, sizeof(int));
    int* max_device;
    hipMalloc(&max_device, sizeof(int));
    int nHilosUsados;
    if (numBloques * numHilos > inputSize)
    {
        nHilosUsados = inputSize;
    }
    else {
        nHilosUsados = numBloques * numHilos;
    }

    int* findmin;
    hipError_t cudaStatusMalloc = hipMalloc(&findmin, nHilosUsados * sizeof(int));
    if (cudaStatusMalloc != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (Array: findmin)");
        hipFree(findmin);
        return 1;
    }
    int* findmax;
    cudaStatusMalloc = hipMalloc(&findmax, nHilosUsados * sizeof(int));
    if (cudaStatusMalloc != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (Array: findmax)");
        hipFree(findmax);
        return 1;
    }

    findMinMax << <numBloques, numHilos >> > (input_device, inputSize, min_device, max_device, findmin, findmax);
    hipDeviceSynchronize();
    calcularMinMax << <numBloques, numHilos >> > ( inputSize, min_device, max_device, findmin, findmax);
    hipDeviceSynchronize();

    int* min = (int*)malloc(sizeof(int));
    hipError_t cudaStatusMemcpy = hipMemcpy(min, min_device, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatusMemcpy != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! (Array: min_device)");
        hipFree(min_device);
        return 1;
    }
    int* max = (int*)malloc(sizeof(int));
    cudaStatusMemcpy = hipMemcpy(max, max_device, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatusMemcpy != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! (Array: max_device)");
        hipFree(max_device);
        return 1;
    }

    *minimo = *min;
    *maximo = *max;

    return 0;
}

__global__ void inicializamosOcurrencias(int* ocurrencias, int ocurrenciasSize) {
    int idHilo = threadIdx.x + blockIdx.x * blockDim.x;
    int nHilosTotal = blockDim.x * gridDim.x;

    //INICIALIZAMOS OCURRENCIAS
    int nhilosInicializaOcurrencias;
    if (nHilosTotal > ocurrenciasSize)
    {
        nhilosInicializaOcurrencias = ocurrenciasSize;
    }
    else {
        nhilosInicializaOcurrencias = nHilosTotal;
    }

    if (idHilo < nhilosInicializaOcurrencias) {

        int nElementosAcalcular = (ocurrenciasSize / nhilosInicializaOcurrencias);
        if ((ocurrenciasSize % nhilosInicializaOcurrencias) > idHilo)
        {
            nElementosAcalcular += 1;
        }

        for (int i = 0; i < nElementosAcalcular; i++)
        {
            int posOcurr = idHilo + (nhilosInicializaOcurrencias * i);
            ocurrencias[posOcurr] = 0;

        }
    }
}

__global__ void contarOcurrencias(int* input, int inputSize, int* ocurrencias, int ocurrenciasSize, int min, int max) {
    int idHilo = threadIdx.x + blockIdx.x * blockDim.x;
    int nHilosTotal = blockDim.x * gridDim.x;

    //CONTAMOS OCURRENCIAS VERTICAL
    int nHilosUsados;
    if (nHilosTotal > inputSize)
    {
        nHilosUsados = inputSize;
    }
    else {
        nHilosUsados = nHilosTotal;
    }
    if (idHilo < nHilosUsados) {

        int nPosASumar = (inputSize / nHilosUsados);
        if (idHilo < (inputSize % nHilosUsados)) { nPosASumar++; }
        for (int i = 0; i < nPosASumar; i++)
        {
            int valorInput = input[idHilo + (i * nHilosUsados)];
            int posEnMinMax = valorInput - min;
            ocurrencias[idHilo + (nHilosUsados * posEnMinMax)] += 1;
        }
    }
}

__global__ void sumarOcurrencias(int inputSize, int* ocurrencias, int ocurrenciasSize, int* aux, int* auxsumado, int min, int max) {
    int idHilo = threadIdx.x + blockIdx.x * blockDim.x;
    int nHilosTotal = blockDim.x * gridDim.x;

    int nHilosUsados;
    if (nHilosTotal > inputSize)
    {
        nHilosUsados = inputSize;
    }
    else {
        nHilosUsados = nHilosTotal;
    }
    //SUMAMOS COLUMNAS OCURRENCIA
    int nHilosUsadosSumaOcurrencias;
    int minmaxSize = max + 1 - min;
    if (nHilosTotal > minmaxSize)
    {
        nHilosUsadosSumaOcurrencias = minmaxSize;
    }
    else {
        nHilosUsadosSumaOcurrencias = nHilosTotal;
    }

    if (idHilo < nHilosUsadosSumaOcurrencias) {

        int nfilasAcalcular = (minmaxSize / nHilosUsadosSumaOcurrencias);
        if ((minmaxSize % nHilosUsadosSumaOcurrencias) > idHilo)
        {
            nfilasAcalcular += 1;
        }

        for (int i = 0; i < nfilasAcalcular; i++)
        {
            for (int j = 0; j < nHilosUsados; j++)
            {
                aux[idHilo + (nHilosUsadosSumaOcurrencias * i)] += ocurrencias[(((idHilo + (nHilosUsadosSumaOcurrencias * i)) * nHilosUsados) + j)];
            }
        }

    }
}

__global__ void calcularAuxEscalera(int* aux, int* auxsumado, int min, int max) {
    int idHilo = threadIdx.x + blockIdx.x * blockDim.x;

    //CALCULAMOS AUXESCALERA
    if (idHilo == 0) {

        auxsumado[0] = aux[0];
        for (int i = 1; i < max + 1 - min; i++)
        {
            auxsumado[i] = aux[i] + auxsumado[i - 1];
        }

    }
}

__global__ void crearSorted1(int* input, int inputSize, int* aux, int* auxsumado, int* sorted, int min, int max) {

    int idHilo = threadIdx.x + blockIdx.x * blockDim.x;
    int nHilosTotal = blockDim.x * gridDim.x;
    int minmaxSize = max + 1 - min;
    //CREAMOS EL ARRAY SORTED
    int nHilosUsadosSorting;
    if (nHilosTotal > minmaxSize)
    {
        nHilosUsadosSorting = minmaxSize;
    }
    else {
        nHilosUsadosSorting = nHilosTotal;
    }

    if (idHilo < nHilosUsadosSorting) {

        int nfilasAcalcular = (minmaxSize / nHilosUsadosSorting);
        if ((minmaxSize % nHilosUsadosSorting) > idHilo)
        {
            nfilasAcalcular += 1;
        }

        for (int i = 0; i < nfilasAcalcular; i++)
        {

            int posAux = idHilo + (nHilosUsadosSorting * i);
            int valor = min + posAux;

            for (int j = 0; j < aux[posAux]; j++)
            {
                sorted[(auxsumado[posAux] - aux[posAux]) + j] = valor;
            }
        }
    }
}

__global__ void crearSorted2(int* input, int inputSize, int* aux, int* auxsumado, int* sorted, int* indicesNuevos, int min, int max) {

    int idHilo = threadIdx.x + blockIdx.x * blockDim.x;
    int nHilosTotal = blockDim.x * gridDim.x;
    int minmaxSize = max + 1 - min;
    //CREAMOS EL ARRAY SORTED
    int nHilosUsadosSorting;
    if (nHilosTotal > inputSize)
    {
        nHilosUsadosSorting = inputSize;
    }
    else {
        nHilosUsadosSorting = nHilosTotal;
    }

    if (idHilo < nHilosUsadosSorting) {

        int nfilasAcalcular = (inputSize / nHilosUsadosSorting);
        if ((inputSize % nHilosUsadosSorting) > idHilo)
        {
            nfilasAcalcular += 1;
        }

        for (int i = 0; i < nfilasAcalcular; i++)
        {

            int num_A_ordenar = input[idHilo + (nHilosUsadosSorting * i)];
            int posEnMinMax = num_A_ordenar - min;
            int posEnAux = posEnMinMax;
            int posEnSorted = atomicSub(&auxsumado[posEnAux], 1);
            posEnSorted--;
            sorted[posEnSorted] = num_A_ordenar;
            indicesNuevos[idHilo + (nHilosUsadosSorting * i)] = posEnSorted;
        }
    }


}

//METODO PRINCIPAL
int CountingSortParalelo(int* input, int inputSize, int numBloques, int numHilos, char metodoCrearSort) {

    hipError_t cudaStatusMalloc;
    hipError_t cudaStatusMemcpy;

    //INICIAMOS CRONOMETRO TIEMPO
    hipEvent_t inicio, fin;
    hipEventCreate(&inicio);
    hipEventCreate(&fin);
    hipEventRecord(inicio, 0);

    //PASAMOS INPUT A LA GRAFICA 
    int* input_device;
    cudaStatusMalloc = hipMalloc(&input_device, inputSize * sizeof(int));
    if (cudaStatusMalloc != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (Array: INPUT)");
        hipFree(input_device);
        goto Error;
    }
    cudaStatusMemcpy = hipMemcpy(input_device, input, inputSize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatusMemcpy != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! (Array: INPUT)");
        hipFree(input_device);
        goto Error;
    }

    //RECOJEMOS EL VALOR MIN Y EL VALOR MAX QUE APARECE EN EL ARRAY
    int minimo;
    int maximo;
    int toret = findMinMaxDeInput(input_device, inputSize, &minimo, &maximo, numBloques, numHilos);
    if (toret == 1) {
        goto Error;
    }
    //CREAMOS EL ARRAY DE OCURRENCIAS  
    int ocurrenciasSize;
    if (numBloques * numHilos > inputSize)
    {
        ocurrenciasSize = inputSize;
    }
    else {
        ocurrenciasSize = numBloques * numHilos;
    }

    ocurrenciasSize *= ((maximo + 1) - minimo);

    ////Pasamos el array OCURRENCIAS a la GPU
    int* ocurrencias_device;
    cudaStatusMalloc = hipMalloc(&ocurrencias_device, ocurrenciasSize * sizeof(int));
    if (cudaStatusMalloc != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!(Array: Ocurrencias)");
        hipFree(ocurrencias_device);
        goto Error;
    }

    //CREAMOS EL ARRAY AUXILIAR Y LO PASAMOS LA GRAFICA
    int* aux_device;
    int aux_size = (maximo + 1) - minimo;
    cudaStatusMalloc = hipMalloc(&aux_device, aux_size * sizeof(int));
    if (cudaStatusMalloc != hipSuccess) {
        fprintf(stderr, "hipMalloc  failed! (Array: Aux)");
        hipFree(aux_device);
        goto Error;
    }

    //CREAMOS EL ARRAY ESCALERA Y LO PASAMOS LA GRAFICA
    int* auxEscalera_device;
    cudaStatusMalloc = hipMalloc(&auxEscalera_device, aux_size * sizeof(int));
    if (cudaStatusMalloc != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (Array: AuxEscalera)");
        hipFree(auxEscalera_device);
        goto Error;
    }

    //CREAMOS EL ARRAY SORTED Y LO PASAMOS LA GRAFICA
    int* sorted_device;
    int sorted_size = inputSize;
    cudaStatusMalloc = hipMalloc(&sorted_device, sorted_size * sizeof(int));
    if (cudaStatusMalloc != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! (Array: Sorted)");
        hipFree(sorted_device);
        goto Error;
    }

    //CREAMOS EL ARRAY INDICES_NUEVOS Y LO PASAMOS LA GRAFICA
    int* indicesNuevos_device;
    if (metodoCrearSort == '2') {
        int indicesNuevos_size = inputSize;
        cudaStatusMalloc = hipMalloc(&indicesNuevos_device, indicesNuevos_size * sizeof(int));
        if (cudaStatusMalloc != hipSuccess) {
            fprintf(stderr, "hipMalloc failed! (Array: indicesNuevos)");
            hipFree(indicesNuevos_device);
            goto Error;
        }
    }

    //ORDENAMOS DE FORMA PARALELA
    inicializamosOcurrencias << <numBloques, numHilos >> > (ocurrencias_device, ocurrenciasSize);
    hipDeviceSynchronize();
    contarOcurrencias << <numBloques, numHilos >> > (input_device, inputSize, ocurrencias_device, ocurrenciasSize, minimo, maximo);
    hipDeviceSynchronize();
    sumarOcurrencias << <numBloques, numHilos >> > (inputSize, ocurrencias_device, ocurrenciasSize, aux_device, auxEscalera_device, minimo, maximo);
    hipDeviceSynchronize();
    calcularAuxEscalera << <numBloques, numHilos >> > (aux_device, auxEscalera_device, minimo, maximo);
    hipDeviceSynchronize();

    if (metodoCrearSort == '1') {
        crearSorted1 << <numBloques, numHilos >> > (input_device, inputSize, aux_device, auxEscalera_device, sorted_device, minimo, maximo);
        hipDeviceSynchronize();
    }
    else {
        crearSorted2 << <numBloques, numHilos >> > (input_device, inputSize, aux_device, auxEscalera_device, sorted_device, indicesNuevos_device, minimo, maximo);
        hipDeviceSynchronize();
    }

    //PARAMOS CRONOMETRO
    hipEventRecord(fin, 0);
    hipEventSynchronize(fin);
    float tiempo;
    hipEventElapsedTime(&tiempo, inicio, fin);

    //RECOJEMOS AUX POR SI LO NECESITA
    int* aux = (int*)malloc(aux_size * sizeof(int));
    cudaStatusMemcpy = hipMemcpy(aux, aux_device, aux_size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatusMemcpy != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! (Array: Aux)");
        hipFree(aux_device);
        goto Error;
    }
    
    //RECOGEMOS EL RESULTADO de SORTED_DEVICE TRAS LA EJECUCION DEL KERNEL
    int* sorted = (int*)malloc(sorted_size * sizeof(int));
    cudaStatusMemcpy = hipMemcpy(sorted, sorted_device, sorted_size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatusMemcpy != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! (Array: Sorted)");
        hipFree(sorted_device);
        goto Error;
    }

    int* indicesNuevos;
    if (metodoCrearSort != '1') {
        indicesNuevos = (int*)malloc(sorted_size * sizeof(int));
        cudaStatusMemcpy = hipMemcpy(indicesNuevos, indicesNuevos_device, sorted_size * sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatusMemcpy != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed! (Array: indicesNuevos)");
            hipFree(indicesNuevos_device);
            goto Error;
        }
    }


    //TESTS 
    printf("Testeando... ");
    int testResult = testerOrdenadoParalelo(sorted_device, inputSize, numBloques, numHilos);
    if (testResult == 0) {
        printf("Ordenado Paralelo Correcto\n");
    }
    else if(testResult == 1){
        printf("Ordenado Paralelo NO Correcto\n");
    }
    else {
        goto Error;
    }

    //MOSTRAR DATOS
    printf("Bloques,Hilos: %d,%d\t", numBloques, numHilos);
    printf("Tamano array: %d\t", inputSize);
    printf("Valores: %d, [%d,%d]\t", maximo + 1 - minimo, minimo, maximo);
    printf("Tiempo tardado: %f milisegundos\n\n", tiempo);

    char option;
    printf("¿Quieres ver el array input? (s/n): ");
    scanf(" %c", &option);

    if (option == 's' || option == 'S') {
        mostrarArray(input, inputSize);
        printf("\n");
    }

    if (metodoCrearSort != '1') {
        printf("¿Quieres ver el array indicesNuevos? (s/n): ");
        scanf(" %c", &option);

        if (option == 's' || option == 'S') {
            mostrarArray(indicesNuevos, inputSize);
            printf("\n");
        }
    }

    printf("¿Quieres ver el array sorted? (s/n): ");
    scanf(" %c", &option);

    if (option == 's' || option == 'S') {
        mostrarArray(sorted, inputSize);
        printf("\n");
    }

    printf("¿Quieres ver el array aux? (s/n): ");
    scanf(" %c", &option);

    if (option == 's' || option == 'S') {
        mostrarArray(aux, aux_size);
        printf("\n");
    }
    

    //LIBERAMOS MEMORIA
    free(aux);
    free(sorted);
    if (metodoCrearSort != '1') { free(indicesNuevos); }
    hipFree(input_device);
    hipFree(aux_device);
    hipFree(auxEscalera_device);
    hipFree(ocurrencias_device);
    hipFree(sorted_device);
    if (metodoCrearSort != '1') { hipFree(indicesNuevos_device);}

    hipError_t cudaStatusFinal = hipDeviceReset();
    if (cudaStatusFinal != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
    //FIN

    //MANEJO DE ERRORES
Error:
    
    fprintf(stderr, "\nSe ha producido algun error\nSeguramente al reservar memoria debido al gran tamano de algun array\nReseteando device...\n\n");
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return -1;
}

void prueba1() {
    int size = 10;
    int* input = generarArrayAleatorio(size, 1, size);

    CountingSortParalelo(input, size, 1, 3, '1');
    CountingSortParalelo(input, size, 1, 3, '2');
}

void prueba2() {
    //100
    int size100 = 100;
    int* input100 = generarArrayAleatorio(size100, 1, size100);
    CountingSortParalelo(input100, size100, 32, 32, '1');
    //500
    int size500 = 500;
    int* input500 = generarArrayAleatorio(size500, 1, size500);
    CountingSortParalelo(input500, size500, 32, 32, '2');
    //1000
    int size1000 = 1000;
    int* input1000 = generarArrayAleatorio(size1000, 1, size1000);
    CountingSortParalelo(input1000, size1000, 32, 32, '1');
    //10000
    int size10000 = 10000;
    int* input10000 = generarArrayAleatorio(size10000, 1, size10000);
    CountingSortParalelo(input10000, size10000, 32, 32, '2');
    //50000
    int size50000 = 50000;
    int* input50000 = generarArrayAleatorio(size50000, 1, size50000);
    CountingSortParalelo(input50000, size50000, 32, 32, '1');
    //200000
    int size200000 = 200000;
    int* input200000 = generarArrayAleatorio(size200000, 1, size200000);
    CountingSortParalelo(input200000, size200000, 32, 32, '2');
    //500000
     int size500000 = 500000;
    int* input500000 = generarArrayAleatorio(size500000, 1, size500000);
    CountingSortParalelo(input500000, size500000, 32, 32, '1');
}

void prueba3() {
    
    //Size demasiado grande
    int sizeDemasiadoGrande = 1000000000;
    int* inputDemasiadoGrande = generarArrayAleatorio(sizeDemasiadoGrande, 1, 10000);
    CountingSortParalelo(inputDemasiadoGrande, sizeDemasiadoGrande, 32, 32, '1');

    //Size viable
    int sizeViable = 100;
    int* inputViable = generarArrayAleatorio(sizeViable, 1, 10000);
    CountingSortParalelo(inputViable, sizeViable, 32, 32, '1');

}

void prueba4() {
    
    int size = 50000000;
    int* input = generarArrayAleatorio(size, 1, 65535);
    CountingSortParalelo(input, size, 1, 1, '1');
    CountingSortParalelo(input, size, 1, 2, '1');
    CountingSortParalelo(input, size, 1, 4, '1');
    CountingSortParalelo(input, size, 1, 8, '1');
    CountingSortParalelo(input, size, 1, 16, '1');
    CountingSortParalelo(input, size, 1, 32, '1');
    CountingSortParalelo(input, size, 1, 64, '1');
    CountingSortParalelo(input, size, 1, 128, '1');
    CountingSortParalelo(input, size, 1, 256, '1');
    CountingSortParalelo(input, size, 1, 512, '1');
    CountingSortParalelo(input, size, 1, 1024, '1');
    CountingSortParalelo(input, size, 5, 256, '1');
    CountingSortParalelo(input, size, 6, 256, '1');
    CountingSortParalelo(input, size, 7, 256, '1');
    CountingSortParalelo(input, size, 2, 1024, '1');
}


int main(int argc, char* argv[]) {

    //PRESENTACION
    presentacion();
    
    int size = 10;
    int* input = generarArrayAleatorio(size, 1, 65535);
    int numBloques = 1;
    int numHilosPorBloque = 32;
    char metodoCreaSorted = '1';

    CountingSortParalelo(input, size, numBloques, numHilosPorBloque, metodoCreaSorted);
        
    //ANTES DE EJECUTAR LAS PRUEBAS, COMENTA LOS PRINTS DE SALIDA LINEAS [657,670]
    //prueba1();
    //prueba2();
    //prueba3();
    //prueba4();

    return 0;
}